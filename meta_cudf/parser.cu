//#include "opt1/meta_def.cuh"
#include <fstream>
#include <memory>

#include <boost/mp11.hpp>
#include <cudf/io/types.hpp>
#include <thrust/logical.h>
#include <iomanip>
#include <meta_json_parser/parser_output_device.cuh>
#include <meta_json_parser/parser_kernel.cuh>
#include <meta_json_parser/action/jstring.cuh>

#include <meta_def.cuh>

using namespace std;
using namespace boost::mp11;

hipStream_t stream;

enum class end_of_line {
    unknown,
    uniks, //< LF, or "\n": end-of-line convention used by Unix
    win   //< CRLF, or "\r\n": end-of-line convention used by MS Windows
};

namespace EndOfLine
{
    struct Unix {};
    struct Win {};
}

struct NoError
{
    __device__ __host__ bool operator()(ParsingError e)
    {
        return ParsingError::None == e;
    }
};

template<class EndOfLineT>
struct LineEndingHelper
{
private:
    __device__ __forceinline__ static void error() { assert("Unknown end of line."); }
public:
    __device__ __forceinline__ static uint32_t get_mask(const uint32_t& val) { error(); return 0; }
    __device__ __forceinline__ static bool is_newline(const uint32_t& val) { error(); return false; }
    __device__ __forceinline__ static uint32_t eol_length() { error(); return 0; }
};

template<>
struct LineEndingHelper<EndOfLine::Unix>
{
    __device__ __forceinline__ static uint32_t get_mask(const uint32_t& val)
    {
        return __vcmpeq4(val, '\n\n\n\n');
    }
    __device__ __forceinline__ static bool is_newline(const uint32_t& val)
    {
        return get_mask(val);
    }
    __device__ __forceinline__ static constexpr uint32_t eol_length()
    {
        return 1;
    }
};

/// <summary>
/// Implemented with assumption that \r can only be found right before \n
/// </summary>
template<>
struct LineEndingHelper<EndOfLine::Win>
{
    __device__ __forceinline__ static uint32_t get_mask(const uint32_t& val)
    {
        return __vcmpeq4(val, '\r\r\r\r');
    }
    __device__ __forceinline__ static bool is_newline(const uint32_t& val)
    {
        return get_mask(val);
    }
    __device__ __forceinline__ static constexpr uint32_t eol_length()
    {
        return 2;
    }
};

template<class EndOfLineT>
struct IsNewLine
{
    __device__ __forceinline__ bool operator()(const hipcub::KeyValuePair<ptrdiff_t, uint32_t> c) const {
        return LineEndingHelper<EndOfLineT>::is_newline(c.value);
    }
};

template<class EndOfLineT>
class OutputIndicesIterator
{
public:

    // Required iterator traits
    typedef OutputIndicesIterator<EndOfLineT>            self_type;              ///< My own type
    typedef ptrdiff_t                                    difference_type;        ///< Type to express the result of subtracting one iterator from another
    typedef hipcub::KeyValuePair<difference_type, uint32_t> value_type;             ///< The type of the element the iterator can point to
    typedef value_type*                                  pointer;                ///< The type of a pointer to an element the iterator can point to
    typedef value_type                                   reference;              ///< The type of a reference to an element the iterator can point to

#if (THRUST_VERSION >= 100700)
    // Use Thrust's iterator categories so we can use these iterators in Thrust 1.7 (or newer) methods
    typedef typename thrust::detail::iterator_facade_category<
        thrust::any_system_tag,
        thrust::random_access_traversal_tag,
        value_type,
        reference
    >::type iterator_category;                                        ///< The iterator category
#else
    typedef std::random_access_iterator_tag     iterator_category;      ///< The iterator category
#endif  // THRUST_VERSION

private:

    InputIndex*  itr;

public:

    /// Constructor
    __host__ __device__ __forceinline__ OutputIndicesIterator(InputIndex* itr) : itr(itr) {}

    /// Assignment operator
    __device__ __forceinline__ self_type& operator=(const value_type &val)
    {
        int inner_offset = LineEndingHelper<EndOfLineT>::eol_length();
        //undefined behavior for 2 byte jsons. e.g. \n[]\n or \n{}\n
        uint32_t mask = LineEndingHelper<EndOfLineT>::get_mask(val.value);
        switch (mask)
        {
            case 0xFF'00'00'00u:
                inner_offset += 3;
                break;
            case 0x00'FF'00'00u:
                inner_offset += 2;
                break;
            case 0x00'00'FF'00u:
                inner_offset += 1;
                break;
            case 0x00'00'00'FFu:
                //inner_offset += 0;
                break;
            default:
                break;
        }
        *itr = static_cast<InputIndex>(val.key * 4) + inner_offset;
        return *this;
    }

    /// Array subscript
    template <typename Distance>
    __host__ __device__ __forceinline__ self_type operator[](Distance n)
    {
        self_type offset = OutputIndicesIterator(itr + n);
        return offset;
    }
};

struct benchmark_input
{
    vector<char> data;
    int count;
    end_of_line eol;
    int bytes_per_string;
};

struct benchmark_device_buffers
{
    ParserOutputDevice<BaseAction> parser_output_buffers;
    char* readonly_buffers;
    char* input_buffer;
    InputIndex* indices_buffer;
    ParsingError* err_buffer;
    void** output_buffers;
    int count;

    vector<void*> host_output_buffers;
};

benchmark_input get_input(const char* filename, int input_count);
KernelLaunchConfiguration prepare_dynamic_config(benchmark_input& input);
benchmark_device_buffers initialize_buffers(benchmark_input& input, KernelLaunchConfiguration* conf);
end_of_line detect_eol(benchmark_input& input);
void launch_kernel(benchmark_device_buffers& device_buffers);

template<class EndOfLineT>
void find_newlines(char* d_input, size_t input_size, InputIndex* d_indices, int count)
{
    InputIndex just_zero = 0;
    hipMemcpyAsync(d_indices, &just_zero, sizeof(InputIndex), hipMemcpyHostToDevice, stream);

    hipcub::ArgIndexInputIterator<uint32_t*> arg_iter(reinterpret_cast<uint32_t*>(d_input));
    OutputIndicesIterator<EndOfLineT> out_iter(d_indices + 1); // +1, we need to add 0 at index 0

    int* d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    int* d_num_selected;
    hipMalloc(&d_num_selected, sizeof(int));

    hipcub::DeviceSelect::If(
        d_temp_storage,
        temp_storage_bytes,
        arg_iter,
        out_iter,
        d_num_selected,
        (input_size + 3) / 4,
        IsNewLine<EndOfLineT>(),
        stream
    );

    hipMalloc(&d_temp_storage, temp_storage_bytes);

    hipcub::DeviceSelect::If(
        d_temp_storage,
        temp_storage_bytes,
        arg_iter,
        out_iter,
        d_num_selected,
        (input_size + 3) / 4,
        IsNewLine<EndOfLineT>(),
        stream
    );

    // Following lines could be commented out as it is only validation step
    hipStreamSynchronize(stream);
    int h_num_selected = -1;
    hipMemcpy(&h_num_selected, d_num_selected, sizeof(int), hipMemcpyDeviceToHost);
    if (h_num_selected != count)
    {
        cout << "Found " << h_num_selected << " new lines instead of declared " << count << ".\n";
        throw runtime_error("Invalid number of new lines.");
    }

    hipFree(d_temp_storage);
    hipFree(d_num_selected);
}

cudf::io::table_with_metadata generate_example_metadata(const char* filename, int count) {
	hipStreamCreate(&stream);

    auto input = get_input(filename, count);

    KernelLaunchConfiguration conf = prepare_dynamic_config(input);
    benchmark_device_buffers device_buffers = initialize_buffers(input, &conf);
    launch_kernel(device_buffers);
    auto cudf_table  = device_buffers.parser_output_buffers.ToCudf(stream);

    vector<string> column_names(cudf_table.num_columns());

    generate(column_names.begin(), column_names.end(), [i = 1]() mutable {
        return "Column " + to_string(i++);
    });

    cudf::io::table_metadata metadata;
    std::for_each(begin(column_names), end(column_names), [&](auto& elem){metadata.schema_info.push_back({elem});});

    return cudf::io::table_with_metadata{
        make_unique<cudf::table>(cudf_table),
        metadata
    };
}

void launch_kernel(benchmark_device_buffers& device_buffers)
{
    using GroupSize = WorkGroupSize;
    constexpr int GROUP_SIZE = WorkGroupSize::value;
    constexpr int GROUP_COUNT = 1024 / GROUP_SIZE;
    using GroupCount = mp_int<GROUP_COUNT>;

    using RT = RuntimeConfiguration<GroupSize, GroupCount>;
    using PC = ParserConfiguration<RT, BaseAction>;
    using PK = ParserKernel<PC>;

    PK pk(device_buffers.parser_output_buffers.m_launch_config, stream);

    pk.Run(
        device_buffers.input_buffer,
        device_buffers.indices_buffer,
        device_buffers.err_buffer,
        device_buffers.output_buffers,
        device_buffers.count,
        device_buffers.host_output_buffers.data()
    );
}

end_of_line detect_eol(benchmark_input& input)
{
    auto found = std::find_if(input.data.begin(), input.data.end(), [](char& c) {
        return c == '\r' || c == '\n';
    });
    if (found == input.data.end())
        return end_of_line::unknown;
    if (*found == '\n')
        return end_of_line::uniks;
    // *found == '\r'
    if ((found + 1) == input.data.end() || *(found + 1) != '\n')
        return end_of_line::unknown;
    return end_of_line::win;
}

KernelLaunchConfiguration prepare_dynamic_config(benchmark_input& input)
{
    KernelLaunchConfiguration conf;

    using DynamicStringActions = mp_copy_if_q<
        ActionIterator<BaseAction>,
        mp_bind<
            mp_similar,
            JStringDynamicCopy<void>,
            _1
        >
    >;

    using DynamicStringActionsV2 = mp_copy_if_q<
        ActionIterator<BaseAction>,
        mp_bind<
            mp_similar,
            JStringDynamicCopyV2<void>,
            _1
        >
    >;

    using DynamicStringActionsV3 = mp_copy_if_q<
        ActionIterator<BaseAction>,
        mp_bind<
            mp_similar,
            JStringDynamicCopyV3<void>,
            _1
        >
    >;

    mp_for_each<
        mp_append<
            DynamicStringActions,
            DynamicStringActionsV2
        >
    >([&conf, &input](auto a) {
        using Action = decltype(a);
        using Tag = typename Action::DynamicStringRequestTag;
        conf.SetDynamicSize<BaseAction, Tag>(input.bytes_per_string);
    });

    mp_for_each<DynamicStringActionsV3>([&conf, &input](auto a) {
        using Action = decltype(a);
        using TagInternal = typename Action::DynamicStringInternalRequestTag;
        conf.SetDynamicSize<BaseAction, TagInternal>(input.bytes_per_string);
        using Tag = typename Action::DynamicStringRequestTag;
        conf.SetDynamicSize<BaseAction, Tag>(input.bytes_per_string);
    });

    return std::move(conf);
}

benchmark_device_buffers initialize_buffers(benchmark_input& input, KernelLaunchConfiguration* conf)
{
    using GroupSize = WorkGroupSize;
    constexpr int GROUP_SIZE = WorkGroupSize::value;
    constexpr int GROUP_COUNT = 1024 / GROUP_SIZE;
    using GroupCount = mp_int<GROUP_COUNT>;
    using RT = RuntimeConfiguration<GroupSize, GroupCount>;
    using PC = ParserConfiguration<RT, BaseAction>;
    using PK = ParserKernel<PC>;
    using M3 = typename PK::M3;
    using BUF = typename M3::ReadOnlyBuffer;
    using KC = typename PK::KC;
    using OM = typename KC::OM;
    constexpr size_t REQUEST_COUNT = boost::mp11::mp_size<typename OutputConfiguration<BaseAction>::RequestList>::value;

    benchmark_device_buffers result;
    result.count = input.count;
    result.parser_output_buffers = ParserOutputDevice<BaseAction>(conf, result.count);
    hipMalloc(&result.readonly_buffers, sizeof(BUF));
    hipMalloc(&result.input_buffer, input.data.size());
    hipMalloc(&result.indices_buffer, sizeof(InputIndex) * (input.count + 1));
    hipMalloc(&result.err_buffer, sizeof(ParsingError) * input.count);
    hipMalloc(&result.output_buffers, sizeof(void*) * REQUEST_COUNT);

    result.host_output_buffers = vector<void*>(REQUEST_COUNT);
    for (int i = 0; i < REQUEST_COUNT; ++i)
    {
        result.host_output_buffers[i] = result.parser_output_buffers.m_d_outputs[i].data().get();
    }

    hipMemcpyAsync(result.input_buffer, input.data.data(), input.data.size(), hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(result.output_buffers, result.host_output_buffers.data(), sizeof(void*) * REQUEST_COUNT, hipMemcpyHostToDevice, stream);

    //End of line might be passed as an option to the program
    if (input.eol == end_of_line::unknown)
        input.eol = detect_eol(input);

    switch (input.eol)
    {
        case end_of_line::uniks:
            find_newlines<EndOfLine::Unix>
                (result.input_buffer, input.data.size(), result.indices_buffer, input.count);
            break;
        case end_of_line::win:
            find_newlines<EndOfLine::Win>
                (result.input_buffer, input.data.size(), result.indices_buffer, input.count);
            break;
        case end_of_line::unknown:
        default:
            std::cerr << "Unknown end of line character!";
            throw std::runtime_error("Unknown end of line character");
    }

    return result;
}

benchmark_input get_input(const char* filename, int input_count)
{
    ifstream file(filename, ifstream::ate | ifstream::binary);
    if (!file.good())
    {
        cout << "Error reading file \"" << filename << "\".\n";
        throw std::runtime_error("Error reading file.");
    }
    vector<char> data(file.tellg());
    file.seekg(0);
    file.read(data.data(), static_cast<streamsize>(data.size()));

    return benchmark_input
        {
            std::move(data),
            input_count,
            end_of_line::unknown,
            32
        };
}


