#include <gtest/gtest.h>
#include <boost/mp11/integral.hpp>
#include <random>
#include <unordered_set>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/logical.h>
#include <meta_json_parser/config.h>
#include <meta_json_parser/parsing_error.h>
#include <meta_json_parser/memory_configuration.h>
#include <meta_json_parser/runtime_configuration.cuh>
#include <meta_json_parser/parser_configuration.h>
#include <meta_json_parser/parser_kernel.cuh>
#include <meta_json_parser/parser_output_device.cuh>
#include <meta_json_parser/action/jstring.cuh>
#include <meta_json_parser/action/void_action.cuh>

class DISABLED_DynamicOutputTest : public ::testing::Test {
public:
#if _DEBUG
	static constexpr size_t TEST_SIZE = 0x11;
#else
	static constexpr size_t TEST_SIZE = 0x8001;
#endif
};

struct TestContextDynamicStringCopy {
	thrust::host_vector<char> h_input;
	thrust::host_vector<int>  h_correct_offsets;
	thrust::host_vector<char> h_correct_content;
	thrust::host_vector<InputIndex> h_indices;
	thrust::device_vector<char> d_input;
	thrust::device_vector<int>  d_correct_offsets;
	thrust::device_vector<char> d_correct_content;
	thrust::device_vector<InputIndex> d_indices;

	TestContextDynamicStringCopy(size_t testSize, size_t group_size, size_t max_str_len) {
		std::minstd_rand rng;
		std::uniform_int_distribution<uint32_t> r_chars((uint32_t)'A', (uint32_t)'Z');
		const size_t MIN_LEN = 1;
		const size_t MAX_LEN = max_str_len;
		const size_t MAX_STR_LEN = MAX_LEN + 3; //" + " + \0
		std::uniform_int_distribution<uint32_t> r_len(MIN_LEN, MAX_LEN);
		h_input = thrust::host_vector<char>(testSize * MAX_STR_LEN);
		h_indices = thrust::host_vector<InputIndex>(testSize + 1);
		h_correct_offsets = thrust::host_vector<int>(testSize + 1);
		h_correct_content = thrust::host_vector<char>();
		auto inp_it = h_input.data();
		auto ind_it = h_indices.begin();
		auto off_it = h_correct_offsets.begin();
		auto out_it = std::back_inserter(h_correct_content);
		*off_it = 0;
		*ind_it = 0;
		++ind_it;
		std::vector<char> escapable({'"', '\\', '/', 'b', 'f', 'n', 'r', 't' });
		std::vector<char> word(MAX_LEN + 1);
		for (size_t i = 0; i < testSize; ++i)
		{
			std::unordered_set<uint32_t> used_pos;
			auto len = r_len(rng);
			*std::generate_n(word.begin(), len, [&]() { return r_chars(rng); }) = '\0';
			while (len != 1)
			{
				auto slash = r_len(rng) % (len - 1);
				if (used_pos.find(slash) != used_pos.end() || used_pos.find(slash + 1) != used_pos.end())
					break;
				word[slash] = '\\';
				word[slash + 1] = escapable[r_len(rng) % escapable.size()];
				used_pos.insert(slash);
				used_pos.insert(slash + 1);
			}
			auto offset = *off_it + len;
			++off_it;
			*off_it = offset;

			out_it = std::copy_n(word.begin(), len, out_it);
			inp_it += snprintf(inp_it, MAX_STR_LEN, "\"%s\"", word.data());
			*ind_it = (inp_it - h_input.data());
			++ind_it;
		}
		d_input = thrust::device_vector<char>(h_input.size() + 256); //256 to allow batch loading
		thrust::copy(h_input.begin(), h_input.end(), d_input.begin());
		d_indices = thrust::device_vector<InputIndex>(h_indices);
		d_correct_offsets = thrust::device_vector<int>(h_correct_offsets);
		d_correct_content = thrust::device_vector<char>(h_correct_content);
	}
};

struct no_error {
	typedef bool result_type;
	typedef ParsingError argument_type;

	__host__ __device__ bool operator()(const ParsingError& err)
	{
		return err == ParsingError::None;
	}
};

template<int GroupSizeT>
void templated_DynamicStringCopy()
{
	using GroupSize = boost::mp11::mp_int<GroupSizeT>;
	constexpr int GROUP_SIZE = GroupSizeT;
	constexpr int GROUP_COUNT = 1024 / GROUP_SIZE;
	using GroupCount = boost::mp11::mp_int<GROUP_COUNT>;
	using MC = EmptyMemoryConfiguration;
	using RT = RuntimeConfiguration<GroupSize, GroupCount>;
	using PC = ParserConfiguration<RT, MC>;
	using BA = JStringDynamicCopy<int>;
	using PK = ParserKernel<PC, BA>;
	const size_t INPUT_T = DISABLED_DynamicOutputTest::TEST_SIZE;
	const size_t MAX_LEN = 6;
	TestContextDynamicStringCopy context(INPUT_T, GROUP_SIZE, MAX_LEN);
	const unsigned int BLOCKS_COUNT = (INPUT_T + GROUP_COUNT - 1) / GROUP_COUNT;
	thrust::device_vector<ParsingError> d_err(INPUT_T);
	thrust::fill(d_err.begin(), d_err.end(), ParsingError::Other);
	ASSERT_TRUE(hipDeviceSynchronize() == hipError_t::hipSuccess);

	KernelLaunchConfiguration klc;
	klc.dynamic_sizes.push_back(MAX_LEN);
	ParserOutputDevice<BA> output(&klc, INPUT_T);

	thrust::host_vector<void*> h_outputs(output.output_buffers_count);
	auto d_output_it = output.m_d_outputs;
	for (auto& h_output : h_outputs)
		h_output = d_output_it++->data().get();
	thrust::device_vector<void*> d_outputs(h_outputs);

	PK pk(&klc);
	pk.Run(
		context.d_input.data().get(),
		context.d_indices.data().get(),
		d_err.data().get(),
		d_outputs.data().get(),
		INPUT_T,
		h_outputs.data()
	);
	ASSERT_TRUE(hipGetLastError() == hipError_t::hipSuccess);
	ASSERT_TRUE(hipDeviceSynchronize() == hipError_t::hipSuccess);
	thrust::host_vector<ParsingError> h_err(d_err);
	auto h_output = output.CopyToHost();
	//ASSERT_TRUE(thrust::equal(context.d_correct.begin(), context.d_correct.end(), d_result.begin()));
	ASSERT_TRUE(thrust::all_of(d_err.begin(), d_err.end(), no_error()));
}


TEST_F(DISABLED_DynamicOutputTest, dynamic_output_copy_string_w32) {
	templated_DynamicStringCopy<32>();
}

