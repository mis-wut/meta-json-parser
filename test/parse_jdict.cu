#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <thrust/logical.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <random>
#include <meta_json_parser/parsing_error.h>
#include <meta_json_parser/action/jnumber.cuh>
#include <meta_json_parser/action/jdict.cuh>
#include <meta_json_parser/parser_kernel.cuh>
#include <meta_json_parser/mp_string.h>

class ParseJDictTest : public ::testing::Test {
public:
#if _DEBUG
	static constexpr size_t TEST_SIZE = 0x1;
#else
	static constexpr size_t TEST_SIZE = 0x8001;
#endif
};

template<class Key1T, class OutType1T, class Key2T, class OutType2T>
struct TestContextJDict2UInt {
	thrust::host_vector<OutType1T> h_correct_1;
	thrust::host_vector<OutType2T> h_correct_2;
	thrust::host_vector<char> h_input;
	thrust::host_vector<InputIndex> h_indices;
	thrust::device_vector<OutType1T> d_correct_1;
	thrust::device_vector<OutType2T> d_correct_2;
	thrust::device_vector<char> d_input;
	thrust::device_vector<InputIndex> d_indices;

	TestContextJDict2UInt(size_t testSize, size_t group_size)
	{
		using Generate1T = boost::mp11::mp_if_c<sizeof(OutType1T) == 1, uint16_t, OutType1T>;
		using Generate2T = boost::mp11::mp_if_c<sizeof(OutType1T) == 1, uint16_t, OutType1T>;
		Generate1T MAX_VAL_1 = static_cast<Generate1T>(std::numeric_limits<OutType1T>::max() - 1);
		Generate2T MAX_VAL_2 = static_cast<Generate2T>(std::numeric_limits<OutType2T>::max() - 1);
		size_t MAX_KEY_LEN_1 = boost::mp11::mp_size<Key1T>::value + 2;
		size_t MAX_KEY_LEN_2 = boost::mp11::mp_size<Key2T>::value + 2;
		size_t MAX_UINT_LEN_1 = (size_t)std::ceil(std::log10((double)MAX_VAL_1));
		size_t MAX_UINT_LEN_2 = (size_t)std::ceil(std::log10((double)MAX_VAL_2));
		if (MAX_UINT_LEN_1 > group_size - 1)
		{
			MAX_VAL_1 = 1;
			for (int i = 0; i < group_size - 1; ++i)
				MAX_VAL_1 *= 10;
			MAX_VAL_1 -= 1;
			MAX_UINT_LEN_1 = group_size - 1;
		}
		if (MAX_UINT_LEN_2 > group_size - 1)
		{
			MAX_VAL_2 = 1;
			for (int i = 0; i < group_size - 1; ++i)
				MAX_VAL_2 *= 10;
			MAX_VAL_2 -= 1;
			MAX_UINT_LEN_2 = group_size - 1;
		}
		std::minstd_rand rng;
		std::uniform_int_distribution<Generate1T> dist_1(1, MAX_VAL_1);
		std::uniform_int_distribution<Generate2T> dist_2(1, MAX_VAL_2);
		size_t MAX_LEN = MAX_UINT_LEN_1 + MAX_UINT_LEN_2 + MAX_KEY_LEN_1 + MAX_KEY_LEN_2 + 11;
		h_input = thrust::host_vector<char>(testSize * MAX_LEN);
		h_correct_1 = thrust::host_vector<OutType1T>(testSize);
		h_correct_2 = thrust::host_vector<OutType2T>(testSize);
		h_indices = thrust::host_vector<InputIndex>(testSize + 1);
		std::generate(h_correct_1.begin(), h_correct_1.end(), [&dist_1, &rng]() { return static_cast<OutType1T>(dist_1(rng)); });
		std::generate(h_correct_2.begin(), h_correct_2.end(), [&dist_2, &rng]() { return static_cast<OutType2T>(dist_2(rng)); });
		auto inp_it = h_input.data();
		auto ind_it = h_indices.begin();
		*ind_it = 0;
		++ind_it;
		std::vector<char> key_1(boost::mp11::mp_size<Key1T>::value + 1);
		std::vector<char> key_2(boost::mp11::mp_size<Key2T>::value + 1);
		auto key_it = key_1.begin();
		boost::mp11::mp_for_each<Key1T>([&](auto x) {
			*key_it++ = static_cast<char>(decltype(x)::value);
		});
		*key_it = 0;
		key_it = key_2.begin();
		boost::mp11::mp_for_each<Key2T>([&](auto x) {
			*key_it++ = static_cast<char>(decltype(x)::value);
		});
		*key_it = 0;
		for (size_t i = 0; i < testSize; ++i)
		{
			auto x1 = static_cast<long long unsigned int>(h_correct_1[i]);
			auto x2 = static_cast<long long unsigned int>(h_correct_2[i]);
			char* k1 = key_1.data();
			char* k2 = key_2.data();
			if (dist_1(rng) & 0x1)
			{
				std::swap(x1, x2);
				std::swap(k1, k2);
			}
			inp_it += snprintf(inp_it, MAX_LEN + 1, "{ \"%s\": %llu, \"%s\": %llu }", k1, x1, k2, x2);
			*ind_it = (inp_it - h_input.data());
			++ind_it;
		}
		d_input = thrust::device_vector<char>(h_input.size() + 256); //256 to allow batch loading
		thrust::copy(h_input.begin(), h_input.end(), d_input.begin());
		d_correct_1 = thrust::device_vector<OutType1T>(h_correct_1);
		d_correct_2 = thrust::device_vector<OutType2T>(h_correct_2);
		d_indices = thrust::device_vector<InputIndex>(h_indices);
	}
};

struct no_error {
	typedef bool result_type;
	typedef ParsingError argument_type;

	__host__ __device__ bool operator()(const ParsingError& err)
	{
		return err == ParsingError::None;
	}
};

template<class OutType1T, class OutType2T, int GroupSizeT>
void templated_ParseDict2UInt(ParseJDictTest &test)
{
	using GroupSize = boost::mp11::mp_int<GroupSizeT>;
	constexpr int GROUP_SIZE = GroupSizeT;
	constexpr int GROUP_COUNT = 1024 / GROUP_SIZE;
	using GroupCount = boost::mp11::mp_int<GROUP_COUNT>;
	using RT = RuntimeConfiguration<GroupSize, GroupCount>;
	using Key1 = boost::mp11::mp_string<'K', 'e', 'y', ' ', 'n', 'u', 'm', ' ', 'o', 'n', 'e'>;
	using Key2 = boost::mp11::mp_string<'S', 'e', 'c', 'o', 'n', 'd', ' ', 'k', 'e', 'y'>;
	using BA = JDict<boost::mp11::mp_list<
		boost::mp11::mp_list<Key1, JNumber<OutType1T, Key1>>,
		boost::mp11::mp_list<Key2, JNumber<OutType2T, Key2>>
	>>;
	using PC = ParserConfiguration<RT, BA>;
	using PK = ParserKernel<PC>;
	using M3 = typename PK::M3;
	using BUF = typename M3::ReadOnlyBuffer;
	thrust::host_vector<BUF> h_buff(1);
	M3::FillReadOnlyBuffer(h_buff[0], nullptr);
	const size_t INPUT_T = ParseJDictTest::TEST_SIZE;
	TestContextJDict2UInt<Key1, OutType1T, Key2, OutType2T> context(INPUT_T, GROUP_SIZE);
	const unsigned int BLOCKS_COUNT = (INPUT_T + GROUP_COUNT - 1) / GROUP_COUNT;
	thrust::device_vector<BUF> d_buff(h_buff);
	thrust::device_vector<ParsingError> d_err(INPUT_T);
	thrust::device_vector<OutType1T> d_result_1(INPUT_T);
	thrust::device_vector<OutType2T> d_result_2(INPUT_T);
	thrust::host_vector<void*> h_outputs(2);
	h_outputs[0] = d_result_1.data().get();
	h_outputs[1] = d_result_2.data().get();
	thrust::device_vector<void*> d_outputs(h_outputs);
	thrust::fill(d_err.begin(), d_err.end(), ParsingError::None);
	ASSERT_TRUE(hipDeviceSynchronize() == hipError_t::hipSuccess);
	typename PK::Launcher(&_parser_kernel<PC>)(BLOCKS_COUNT)(
		d_buff.data().get(),
		context.d_input.data().get(),
		context.d_indices.data().get(),
		d_err.data().get(),
		d_outputs.data().get(),
		INPUT_T
	);
	ASSERT_TRUE(hipGetLastError() == hipError_t::hipSuccess);
	ASSERT_TRUE(hipDeviceSynchronize() == hipError_t::hipSuccess);
	thrust::host_vector<ParsingError> h_err(d_err);
	thrust::host_vector<OutType1T> h_result_1(d_result_1);
	thrust::host_vector<OutType2T> h_result_2(d_result_2);
	ASSERT_TRUE(thrust::all_of(d_err.begin(), d_err.end(), no_error()));
	ASSERT_TRUE(thrust::equal(context.d_correct_1.begin(), context.d_correct_1.end(), d_result_1.begin()));
	ASSERT_TRUE(thrust::equal(context.d_correct_2.begin(), context.d_correct_2.end(), d_result_2.begin()));
}

TEST_F(ParseJDictTest, uint8_uint32_W32) {
	templated_ParseDict2UInt<uint8_t, uint32_t, 32>(*this);
}

TEST_F(ParseJDictTest, uint8_uint32_W16) {
	templated_ParseDict2UInt<uint8_t, uint32_t, 16>(*this);
}

TEST_F(ParseJDictTest, uint8_uint32_W8) {
	templated_ParseDict2UInt<uint8_t, uint32_t, 8>(*this);
}

TEST_F(ParseJDictTest, uint64_uint16_W32) {
	templated_ParseDict2UInt<uint64_t, uint16_t, 32>(*this);
}

TEST_F(ParseJDictTest, uint64_uint16_W16) {
	templated_ParseDict2UInt<uint64_t, uint16_t, 16>(*this);
}

TEST_F(ParseJDictTest, uint64_uint16_W8) {
	templated_ParseDict2UInt<uint64_t, uint16_t, 8>(*this);
}

TEST_F(ParseJDictTest, uint64_uint64_W32) {
	templated_ParseDict2UInt<uint64_t, uint64_t, 32>(*this);
}

TEST_F(ParseJDictTest, uint64_uint64_W16) {
	templated_ParseDict2UInt<uint64_t, uint64_t, 16>(*this);
}

TEST_F(ParseJDictTest, uint64_uint64_W8) {
	templated_ParseDict2UInt<uint64_t, uint64_t, 8>(*this);
}

TEST_F(ParseJDictTest, uint8_uint8_W32) {
	templated_ParseDict2UInt<uint8_t, uint8_t, 32>(*this);
}

TEST_F(ParseJDictTest, uint8_uint8_W16) {
	templated_ParseDict2UInt<uint8_t, uint8_t, 16>(*this);
}

TEST_F(ParseJDictTest, uint8_uint8_W8) {
	templated_ParseDict2UInt<uint8_t, uint8_t, 8>(*this);
}

