#include <gtest/gtest.h>
#include <thrust/logical.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <random>
#include <meta_json_parser/parsing_error.h>
#include <meta_json_parser/action/jnumber.cuh>
#include <meta_json_parser/action/jstring.cuh>
#include <meta_json_parser/action/jdict.cuh>
#include <meta_json_parser/action/jarray.cuh>
#include <meta_json_parser/parser_kernel.cuh>
#include <meta_json_parser/mp_string.h>

using namespace boost::mp11;

class ParseBookJson : public ::testing::Test {
public:
};

struct no_error {
	typedef bool result_type;
	typedef ParsingError argument_type;

	__host__ __device__ bool operator()(const ParsingError& err)
	{
		return err == ParsingError::None;
	}
};

template<int GroupSizeT>
void templated_ParseBookJson()
{
	const size_t INPUT_T = 0x8001;
	using GroupSize = mp_int<GroupSizeT>;
	constexpr int GROUP_SIZE = GroupSizeT;
	constexpr int GROUP_COUNT = 1024 / GROUP_SIZE;
	using GroupCount = mp_int<GROUP_COUNT>;
	using WGR = WorkGroupReader<GroupSize>;
	using MC = MemoryConfiguration<mp_list<>, mp_list<>, mp_list<>>;
	using RT = RuntimeConfiguration<GroupSize, GroupCount>;
	using PC = ParserConfiguration<RT, MC>;
	//Keys
	using K_books = mp_string<'b', 'o', 'o', 'k', 's'>;
	using K_isbn = mp_string<'i', 's', 'b', 'n'>;
	using K_title = mp_string<'t', 'i', 't', 'l', 'e'>;
	using K_subtitle = mp_string<'s', 'u', 'b', 't', 'i', 't', 'l', 'e'>;
	using K_author = mp_string<'a', 'u', 't', 'h', 'o', 'r'>;
	using K_published = mp_string<'p', 'u', 'b', 'l', 'i', 's', 'h', 'e', 'd'>;
	using K_publisher = mp_string<'p', 'u', 'b', 'l', 'i', 's', 'h', 'e', 'r'>;
	using K_pages = mp_string<'p', 'a', 'g', 'e', 's'>;
	using K_description = mp_string<'d', 'e', 's', 'c', 'r', 'i', 'p', 't', 'i', 'o', 'n'>;
	using K_website = mp_string<'w', 'e', 'b', 's', 'i', 't', 'e'>;
	//Bytes to copy
	constexpr int B_isbn        = 16;
	constexpr int B_title       = 128;
	constexpr int B_subtitle    = 128;
	constexpr int B_author      = 64;
	constexpr int B_published   = 64;
	constexpr int B_publisher   = 64;
	constexpr int B_description = 512;
	constexpr int B_website     = 64;
	using A_book_entry = JDict<mp_list<
		//Pair < Key          , Action          < Action details                     >>
		//                                      | Bytes to copy       | OutputTag    |
		mp_list<K_isbn,        JStringStaticCopy<mp_int<B_isbn       >, K_isbn       >>,
		mp_list<K_title,       JStringStaticCopy<mp_int<B_title      >, K_title      >>,
		mp_list<K_subtitle,    JStringStaticCopy<mp_int<B_subtitle   >, K_subtitle   >>,
		mp_list<K_author,      JStringStaticCopy<mp_int<B_author     >, K_author     >>,
		mp_list<K_published,   JStringStaticCopy<mp_int<B_published  >, K_published  >>,
		mp_list<K_publisher,   JStringStaticCopy<mp_int<B_publisher  >, K_publisher  >>,
		mp_list<K_description, JStringStaticCopy<mp_int<B_description>, K_description>>,
		mp_list<K_website,     JStringStaticCopy<mp_int<B_website    >, K_website    >>,
		//                                      |OutputType           | OutputTag    |
		mp_list<K_pages,       JNumber          <uint32_t,              K_pages      >>
	>>;
	using A_book_array = JArray<mp_list<
		//Pair < Index   , Action      >
		mp_list<mp_int<0>, A_book_entry>
	>>;
	using BA = JDict<mp_list<
		mp_list<K_books, A_book_array>
	>>;
	using PK = ParserKernel<PC, BA>;
	const char* json_format =
		"{\n"
		"\"books\": [\n"
		"{\n"
		"\"isbn\": \"%s\",\n"
		"\"title\": \"%s\",\n"
		"\"subtitle\": \"%s\",\n"
		"\"author\": \"%s\",\n"
		"\"published\": \"%s\",\n"
		"\"publisher\": \"%s\",\n"
		"\"pages\": %u,\n"
		"\"description\": \"%s\",\n"
		"\"website\": \"%s\"\n"
		"}]}";
	const char* json_isbn = "9781593275846";
	const char* json_title = "Eloquent JavaScript, Second Edition";
	const char* json_subtitle = "A Modern Introduction to Programming";
	const char* json_author = "Marijn Haverbeke";
	const char* json_published = "2014-12-14T00:00:00.000Z";
	const char* json_publisher = "No Starch Press";
	const char* json_description =
		"JavaScript lies at the heart of almost every modern\\n"
		"web application, from social apps to the newest browser-based games.\\n"
		"Though simple for beginners to pick up and play with, JavaScript is a\\n"
		"flexible, complex language that you can use to build full-scale\\n"
		"applications.";
	const uint32_t json_pages = 472;
	const char* json_website = "http://eloquentjavascript.net/";
	//input
	const int MAX_JSON_LEN = 1024;
	thrust::host_vector<char> h_input(INPUT_T * MAX_JSON_LEN);
	thrust::host_vector<InputIndex> h_indices(INPUT_T + 1);
	auto inp_it = h_input.data();
	auto ind_it = h_indices.begin();
	*ind_it = 0;
	++ind_it;
	for (int i = 0; i < INPUT_T; ++i)
	{
		inp_it += snprintf(inp_it, MAX_JSON_LEN, json_format,
			json_isbn, json_title, json_subtitle, json_author,
			json_published, json_publisher, json_pages,
			json_description, json_website
		);
		*ind_it = (inp_it - h_input.data());
		++ind_it;
	}
	thrust::device_vector<char> d_input(h_input);
	thrust::device_vector<InputIndex> d_indices(h_indices);
	using M3 = typename PK::M3;
	using BUF = typename M3::ReadOnlyBuffer;
	thrust::host_vector<BUF> h_buff(1);
	M3::FillReadOnlyBuffer(h_buff[0], nullptr);
	const unsigned int BLOCKS_COUNT = (INPUT_T + GROUP_COUNT - 1) / GROUP_COUNT;
	//correct values
	thrust::host_vector<char> h_c_isbn       (INPUT_T * B_isbn,        '\0');
	thrust::host_vector<char> h_c_title      (INPUT_T * B_title,       '\0');
	thrust::host_vector<char> h_c_subtitle   (INPUT_T * B_subtitle,    '\0');
	thrust::host_vector<char> h_c_author     (INPUT_T * B_author,      '\0');
	thrust::host_vector<char> h_c_published  (INPUT_T * B_published,   '\0');
	thrust::host_vector<char> h_c_publisher  (INPUT_T * B_publisher,   '\0');
	thrust::host_vector<char> h_c_description(INPUT_T * B_description, '\0');
	thrust::host_vector<char> h_c_website    (INPUT_T * B_website,     '\0');
	thrust::host_vector<uint32_t> h_c_pages  (INPUT_T);
	for (int i = 0; i < INPUT_T; i++)
	{
		snprintf(h_c_isbn.data()        + i * B_isbn,        B_isbn,        "%s", json_isbn);
		snprintf(h_c_title.data()       + i * B_title,       B_title,       "%s", json_title);
		snprintf(h_c_subtitle.data()    + i * B_subtitle,    B_subtitle,    "%s", json_subtitle);
		snprintf(h_c_author.data()      + i * B_author,      B_author,      "%s", json_author);
		snprintf(h_c_published.data()   + i * B_published,   B_published,   "%s", json_published);
		snprintf(h_c_publisher.data()   + i * B_publisher,   B_publisher,   "%s", json_publisher);
		snprintf(h_c_description.data() + i * B_description, B_description, "%s", json_description);
		snprintf(h_c_website.data()     + i * B_website,     B_website,     "%s", json_website);
		h_c_pages[i] = json_pages;
	}
	//Readonly buffers
	thrust::device_vector<BUF> d_buff(h_buff);
	//Parsing errors
	thrust::device_vector<ParsingError> d_err(INPUT_T);
	//output values
	thrust::device_vector<char> d_r_isbn       (INPUT_T * B_isbn       );
	thrust::device_vector<char> d_r_title      (INPUT_T * B_title      );
	thrust::device_vector<char> d_r_subtitle   (INPUT_T * B_subtitle   );
	thrust::device_vector<char> d_r_author     (INPUT_T * B_author     );
	thrust::device_vector<char> d_r_published  (INPUT_T * B_published  );
	thrust::device_vector<char> d_r_publisher  (INPUT_T * B_publisher  );
	thrust::device_vector<char> d_r_description(INPUT_T * B_description);
	thrust::device_vector<char> d_r_website    (INPUT_T * B_website    );
	thrust::device_vector<uint32_t> d_r_pages  (INPUT_T);
	thrust::host_vector<void*> h_outputs(9);
	h_outputs[PK::KC::OM::template TagIndex<K_isbn>::value       ] = d_r_isbn.data().get();
	h_outputs[PK::KC::OM::template TagIndex<K_title>::value      ] = d_r_title.data().get();
	h_outputs[PK::KC::OM::template TagIndex<K_subtitle>::value   ] = d_r_subtitle.data().get();
	h_outputs[PK::KC::OM::template TagIndex<K_author>::value     ] = d_r_author.data().get();
	h_outputs[PK::KC::OM::template TagIndex<K_published>::value  ] = d_r_published.data().get();
	h_outputs[PK::KC::OM::template TagIndex<K_publisher>::value  ] = d_r_publisher.data().get();
	h_outputs[PK::KC::OM::template TagIndex<K_description>::value] = d_r_description.data().get();
	h_outputs[PK::KC::OM::template TagIndex<K_website>::value    ] = d_r_website.data().get();
	h_outputs[PK::KC::OM::template TagIndex<K_pages>::value      ] = d_r_pages.data().get();
	thrust::device_vector<void*> d_outputs(h_outputs);
	thrust::fill(d_err.begin(), d_err.end(), ParsingError::None);
	ASSERT_TRUE(hipDeviceSynchronize() == hipError_t::hipSuccess);
	typename PK::Launcher(&_parser_kernel<PC, BA>)(BLOCKS_COUNT)(
		d_buff.data().get(),
		d_input.data().get(),
		d_indices.data().get(),
		d_err.data().get(),
		d_outputs.data().get(),
		INPUT_T
	);
	ASSERT_TRUE(hipGetLastError() == hipError_t::hipSuccess);
	ASSERT_TRUE(hipDeviceSynchronize() == hipError_t::hipSuccess);
	thrust::host_vector<ParsingError> h_err(d_err);
	thrust::host_vector<char> h_r_isbn(d_r_isbn);
	thrust::host_vector<char> h_r_title(d_r_title);
	thrust::host_vector<char> h_r_subtitle(d_r_subtitle);
	thrust::host_vector<char> h_r_author(d_r_author);
	thrust::host_vector<char> h_r_published(d_r_published);
	thrust::host_vector<char> h_r_publisher(d_r_publisher);
	thrust::host_vector<char> h_r_description(d_r_description);
	thrust::host_vector<char> h_r_website(d_r_website);
	thrust::host_vector<uint32_t> h_r_pages(d_r_pages);
	ASSERT_TRUE(thrust::all_of(d_err.begin(), d_err.end(), no_error()));
	ASSERT_TRUE(thrust::equal(h_r_isbn.begin(), h_r_isbn.end(), h_c_isbn.begin()));
	ASSERT_TRUE(thrust::equal(h_r_title.begin(), h_r_title.end(), h_c_title.begin()));
	ASSERT_TRUE(thrust::equal(h_r_subtitle.begin(), h_r_subtitle.end(), h_c_subtitle.begin()));
	ASSERT_TRUE(thrust::equal(h_r_author.begin(), h_r_author.end(), h_c_author.begin()));
	ASSERT_TRUE(thrust::equal(h_r_published.begin(), h_r_published.end(), h_c_published.begin()));
	ASSERT_TRUE(thrust::equal(h_r_publisher.begin(), h_r_publisher.end(), h_c_publisher.begin()));
	ASSERT_TRUE(thrust::equal(h_r_description.begin(), h_r_description.end(), h_c_description.begin()));
	ASSERT_TRUE(thrust::equal(h_r_website.begin(), h_r_website.end(), h_c_website.begin()));
	ASSERT_TRUE(thrust::equal(h_r_pages.begin(), h_r_pages.end(), h_c_pages.begin()));
}

TEST_F(ParseBookJson, parsing_book_json_W32) {
	templated_ParseBookJson<32>();
}

TEST_F(ParseBookJson, parsing_book_json_W16) {
	templated_ParseBookJson<16>();
}

TEST_F(ParseBookJson, parsing_book_json_W8) {
	templated_ParseBookJson<8>();
}
